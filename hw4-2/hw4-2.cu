#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <math.h>
#include <omp.h>
#define BLOCK_SIZE 64
#define HALF_BLOCK_SIZE 32
#define inf 1e9

__global__ void Phase_1(int *dist, int Round, int n) {
    const int innerI = threadIdx.y;
    const int innerJ = threadIdx.x;
    const int offset = BLOCK_SIZE * Round;

    __shared__ int C[BLOCK_SIZE][BLOCK_SIZE]; // 2d

    // Every thread read its own value
    // how index: blockIndex (to next diagonal block) + innerBlockIndex (every thread has its own index)
    C[innerI][innerJ] = dist[offset*(n+1) + innerI*n + innerJ];
    C[innerI+HALF_BLOCK_SIZE][innerJ] = dist[offset*(n+1) + (innerI+HALF_BLOCK_SIZE)*n + innerJ];
    C[innerI][innerJ+HALF_BLOCK_SIZE] = dist[offset*(n+1) + innerI*n + innerJ + HALF_BLOCK_SIZE];
    C[innerI+HALF_BLOCK_SIZE][innerJ+HALF_BLOCK_SIZE] = dist[offset*(n+1) + (innerI+HALF_BLOCK_SIZE)*n + innerJ + HALF_BLOCK_SIZE];
    __syncthreads();

    for (int k = 0; k < BLOCK_SIZE; k++) {
        C[innerI][innerJ] = (C[innerI][k] + C[k][innerJ]) < C[innerI][innerJ] ? (C[innerI][k] + C[k][innerJ]) : C[innerI][innerJ];

        C[innerI+HALF_BLOCK_SIZE][innerJ] = (C[innerI+HALF_BLOCK_SIZE][k] + C[k][innerJ]) < C[innerI+HALF_BLOCK_SIZE][innerJ] ? (C[innerI+HALF_BLOCK_SIZE][k] + C[k][innerJ]) : C[innerI+HALF_BLOCK_SIZE][innerJ];

        C[innerI][innerJ+HALF_BLOCK_SIZE] = (C[innerI][k] + C[k][innerJ+HALF_BLOCK_SIZE]) < C[innerI][innerJ+HALF_BLOCK_SIZE] ? (C[innerI][k] + C[k][innerJ+HALF_BLOCK_SIZE]) : C[innerI][innerJ+HALF_BLOCK_SIZE];

        C[innerI+HALF_BLOCK_SIZE][innerJ+HALF_BLOCK_SIZE] = (C[innerI+HALF_BLOCK_SIZE][k] + C[k][innerJ+HALF_BLOCK_SIZE]) < C[innerI+HALF_BLOCK_SIZE][innerJ+HALF_BLOCK_SIZE] ? (C[innerI+HALF_BLOCK_SIZE][k] + C[k][innerJ+HALF_BLOCK_SIZE]) : C[innerI+HALF_BLOCK_SIZE][innerJ+HALF_BLOCK_SIZE];
        __syncthreads(); // TODO: only phase one
    }

    dist[offset*(n+1) + innerI*n + innerJ] = C[innerI][innerJ];
    dist[offset*(n+1) + (innerI+HALF_BLOCK_SIZE)*n + innerJ] = C[innerI+HALF_BLOCK_SIZE][innerJ];
    dist[offset*(n+1) + innerI*n + innerJ + HALF_BLOCK_SIZE] = C[innerI][innerJ+HALF_BLOCK_SIZE];
    dist[offset*(n+1) + (innerI+HALF_BLOCK_SIZE)*n + innerJ + HALF_BLOCK_SIZE] = C[innerI+HALF_BLOCK_SIZE][innerJ+HALF_BLOCK_SIZE];
}


__global__ void Phase_2(int *dist, int Round, int n) {
    const int i = blockIdx.x; // "i" in n block in one row
    if (i == Round) return;

    const int innerI = threadIdx.y;
    const int innerJ = threadIdx.x;
    const int diagonalOffset = BLOCK_SIZE * Round;

    __shared__ int Diagonal[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ int A[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ int B[BLOCK_SIZE][BLOCK_SIZE];
  
    A[innerI][innerJ] = dist[i*BLOCK_SIZE*n + Round*BLOCK_SIZE + innerI*n + innerJ];
    A[innerI+HALF_BLOCK_SIZE][innerJ] = dist[i*BLOCK_SIZE*n + Round*BLOCK_SIZE + (innerI+HALF_BLOCK_SIZE)*n + innerJ];
    A[innerI][innerJ+HALF_BLOCK_SIZE] = dist[i*BLOCK_SIZE*n + Round*BLOCK_SIZE + innerI*n + innerJ + HALF_BLOCK_SIZE];
    A[innerI + HALF_BLOCK_SIZE][innerJ + HALF_BLOCK_SIZE] = dist[i*BLOCK_SIZE*n + Round*BLOCK_SIZE + (innerI+HALF_BLOCK_SIZE)*n + innerJ+HALF_BLOCK_SIZE];

    B[innerI][innerJ] = dist[Round*BLOCK_SIZE*n + i*BLOCK_SIZE + innerI*n + innerJ];
    B[innerI+HALF_BLOCK_SIZE][innerJ] = dist[Round*BLOCK_SIZE*n + i*BLOCK_SIZE + (innerI+HALF_BLOCK_SIZE)*n + innerJ];
    B[innerI][innerJ+HALF_BLOCK_SIZE] = dist[Round*BLOCK_SIZE*n + i*BLOCK_SIZE + innerI*n + innerJ+HALF_BLOCK_SIZE];
    B[innerI + HALF_BLOCK_SIZE][innerJ + HALF_BLOCK_SIZE] = dist[Round*BLOCK_SIZE*n + i*BLOCK_SIZE + (innerI+HALF_BLOCK_SIZE)*n + innerJ+HALF_BLOCK_SIZE];

    Diagonal[innerI][innerJ] = dist[diagonalOffset*(n+1) + innerI*n + innerJ]; // diagonalValue
    Diagonal[innerI+HALF_BLOCK_SIZE][innerJ] = dist[diagonalOffset*(n+1) + (innerI+HALF_BLOCK_SIZE)*n + innerJ]; // diagonalValue
    Diagonal[innerI][innerJ+HALF_BLOCK_SIZE] = dist[diagonalOffset*(n+1) + innerI*n + innerJ+HALF_BLOCK_SIZE]; // diagonalValue
    Diagonal[innerI+HALF_BLOCK_SIZE][innerJ+HALF_BLOCK_SIZE] = dist[diagonalOffset*(n+1) + (innerI+HALF_BLOCK_SIZE)*n + innerJ+HALF_BLOCK_SIZE]; // diagonalValue
  
    __syncthreads();

    #pragma unroll 32
    for (int k = 0; k < BLOCK_SIZE; k++) {

        A[innerI][innerJ] = (A[innerI][k] + Diagonal[k][innerJ]) < A[innerI][innerJ] ? (A[innerI][k] + Diagonal[k][innerJ]) : A[innerI][innerJ];

        A[innerI+HALF_BLOCK_SIZE][innerJ] = (A[innerI+HALF_BLOCK_SIZE][k] + Diagonal[k][innerJ]) < A[innerI+HALF_BLOCK_SIZE][innerJ] ? (A[innerI+HALF_BLOCK_SIZE][k] + Diagonal[k][innerJ]) : A[innerI+HALF_BLOCK_SIZE][innerJ];

        A[innerI][innerJ+HALF_BLOCK_SIZE] = (A[innerI][k] + Diagonal[k][innerJ+HALF_BLOCK_SIZE]) < A[innerI][innerJ+HALF_BLOCK_SIZE] ? (A[innerI][k] + Diagonal[k][innerJ+HALF_BLOCK_SIZE]) : A[innerI][innerJ+HALF_BLOCK_SIZE];
        
        A[innerI+HALF_BLOCK_SIZE][innerJ+HALF_BLOCK_SIZE] = (A[innerI+HALF_BLOCK_SIZE][k] + Diagonal[k][innerJ+HALF_BLOCK_SIZE]) < A[innerI+HALF_BLOCK_SIZE][innerJ+HALF_BLOCK_SIZE] ? (A[innerI+HALF_BLOCK_SIZE][k] + Diagonal[k][innerJ+HALF_BLOCK_SIZE]) : A[innerI+HALF_BLOCK_SIZE][innerJ+HALF_BLOCK_SIZE];

        B[innerI][innerJ] = (Diagonal[innerI][k] + B[k][innerJ]) < B[innerI][innerJ] ? (Diagonal[innerI][k] + B[k][innerJ]) : B[innerI][innerJ];

        B[innerI+HALF_BLOCK_SIZE][innerJ] = (Diagonal[innerI+HALF_BLOCK_SIZE][k] + B[k][innerJ]) < B[innerI+HALF_BLOCK_SIZE][innerJ] ? (Diagonal[innerI+HALF_BLOCK_SIZE][k] + B[k][innerJ]) : B[innerI+HALF_BLOCK_SIZE][innerJ];

        B[innerI][innerJ+HALF_BLOCK_SIZE] = (Diagonal[innerI][k] + B[k][innerJ+HALF_BLOCK_SIZE]) < B[innerI][innerJ+HALF_BLOCK_SIZE] ? (Diagonal[innerI][k] + B[k][innerJ+HALF_BLOCK_SIZE]) : B[innerI][innerJ+HALF_BLOCK_SIZE];
        
        B[innerI+HALF_BLOCK_SIZE][innerJ+HALF_BLOCK_SIZE] = (Diagonal[innerI+HALF_BLOCK_SIZE][k] + B[k][innerJ+HALF_BLOCK_SIZE]) < B[innerI+HALF_BLOCK_SIZE][innerJ+HALF_BLOCK_SIZE] ? (Diagonal[innerI+HALF_BLOCK_SIZE][k] + B[k][innerJ+HALF_BLOCK_SIZE]) : B[innerI+HALF_BLOCK_SIZE][innerJ+HALF_BLOCK_SIZE];
    }

    dist[i*BLOCK_SIZE*n + Round*BLOCK_SIZE + innerI*n + innerJ] = A[innerI][innerJ];
    dist[i*BLOCK_SIZE*n + Round*BLOCK_SIZE + (innerI+HALF_BLOCK_SIZE)*n + innerJ] = A[innerI+HALF_BLOCK_SIZE][innerJ];
    dist[i*BLOCK_SIZE*n + Round*BLOCK_SIZE + innerI*n + innerJ + HALF_BLOCK_SIZE] = A[innerI][innerJ+HALF_BLOCK_SIZE];
    dist[i*BLOCK_SIZE*n + Round*BLOCK_SIZE + (innerI+HALF_BLOCK_SIZE)*n + innerJ+HALF_BLOCK_SIZE] = A[innerI + HALF_BLOCK_SIZE][innerJ + HALF_BLOCK_SIZE];

    dist[Round*BLOCK_SIZE*n + i*BLOCK_SIZE + innerI*n + innerJ] = B[innerI][innerJ];
    dist[Round*BLOCK_SIZE*n + i*BLOCK_SIZE + (innerI+HALF_BLOCK_SIZE)*n + innerJ] = B[innerI+HALF_BLOCK_SIZE][innerJ];
    dist[Round*BLOCK_SIZE*n + i*BLOCK_SIZE + innerI*n + innerJ+HALF_BLOCK_SIZE] = B[innerI][innerJ+HALF_BLOCK_SIZE];
    dist[Round*BLOCK_SIZE*n + i*BLOCK_SIZE + (innerI+HALF_BLOCK_SIZE)*n + innerJ+HALF_BLOCK_SIZE] = B[innerI + HALF_BLOCK_SIZE][innerJ + HALF_BLOCK_SIZE];
}

__global__ void Phase_3(int *dist, int Round, int n, int yOffset) {
    const int j = blockIdx.x;
    const int i = blockIdx.y + yOffset;
    if (i == Round && j == Round) return;

    const int innerI = threadIdx.y;
    const int innerJ = threadIdx.x;

    __shared__ int A[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ int B[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ int C[BLOCK_SIZE][BLOCK_SIZE];
  
    C[innerI][innerJ] = dist[i*BLOCK_SIZE*n + j*BLOCK_SIZE + innerI*n + innerJ];
    C[innerI+HALF_BLOCK_SIZE][innerJ] = dist[i*BLOCK_SIZE*n + j*BLOCK_SIZE + (innerI+HALF_BLOCK_SIZE)*n + innerJ];
    C[innerI][innerJ+HALF_BLOCK_SIZE] = dist[i*BLOCK_SIZE*n + j*BLOCK_SIZE + innerI*n + innerJ+HALF_BLOCK_SIZE];
    C[innerI+HALF_BLOCK_SIZE][innerJ+HALF_BLOCK_SIZE] = dist[i*BLOCK_SIZE*n + j*BLOCK_SIZE + (innerI+HALF_BLOCK_SIZE)*n + innerJ+HALF_BLOCK_SIZE];


    A[innerI][innerJ] = dist[i*BLOCK_SIZE*n + Round*BLOCK_SIZE + innerI*n + innerJ];
    A[innerI+HALF_BLOCK_SIZE][innerJ] = dist[i*BLOCK_SIZE*n + Round*BLOCK_SIZE + (innerI+HALF_BLOCK_SIZE)*n + innerJ];
    A[innerI][innerJ+HALF_BLOCK_SIZE] = dist[i*BLOCK_SIZE*n + Round*BLOCK_SIZE + innerI*n + innerJ + HALF_BLOCK_SIZE];
    A[innerI + HALF_BLOCK_SIZE][innerJ + HALF_BLOCK_SIZE] = dist[i*BLOCK_SIZE*n + Round*BLOCK_SIZE + (innerI+HALF_BLOCK_SIZE)*n + innerJ + HALF_BLOCK_SIZE];

    B[innerI][innerJ] = dist[Round*BLOCK_SIZE*n + j*BLOCK_SIZE + innerI*n + innerJ];
    B[innerI+HALF_BLOCK_SIZE][innerJ] = dist[Round*BLOCK_SIZE*n + j*BLOCK_SIZE + (innerI+HALF_BLOCK_SIZE)*n + innerJ];
    B[innerI][innerJ+HALF_BLOCK_SIZE] = dist[Round*BLOCK_SIZE*n + j*BLOCK_SIZE + innerI*n + innerJ+HALF_BLOCK_SIZE];
    B[innerI + HALF_BLOCK_SIZE][innerJ + HALF_BLOCK_SIZE] = dist[Round*BLOCK_SIZE*n + j*BLOCK_SIZE + (innerI+HALF_BLOCK_SIZE)*n + innerJ+HALF_BLOCK_SIZE];
  
    __syncthreads();

    #pragma unroll 32
    for (int k = 0; k < BLOCK_SIZE; k++) {
        C[innerI][innerJ] = (A[innerI][k] + B[k][innerJ]) < C[innerI][innerJ] ? (A[innerI][k] + B[k][innerJ]) : C[innerI][innerJ];

        C[innerI+HALF_BLOCK_SIZE][innerJ] = (A[innerI+HALF_BLOCK_SIZE][k] + B[k][innerJ]) < C[innerI+HALF_BLOCK_SIZE][innerJ] ? (A[innerI+HALF_BLOCK_SIZE][k] + B[k][innerJ]) : C[innerI+HALF_BLOCK_SIZE][innerJ];

        C[innerI][innerJ+HALF_BLOCK_SIZE] = (A[innerI][k] + B[k][innerJ+HALF_BLOCK_SIZE]) < C[innerI][innerJ+HALF_BLOCK_SIZE] ? (A[innerI][k] + B[k][innerJ+HALF_BLOCK_SIZE]) : C[innerI][innerJ+HALF_BLOCK_SIZE];
        
        C[innerI+HALF_BLOCK_SIZE][innerJ+HALF_BLOCK_SIZE] = (A[innerI+HALF_BLOCK_SIZE][k] + B[k][innerJ+HALF_BLOCK_SIZE]) < C[innerI+HALF_BLOCK_SIZE][innerJ+HALF_BLOCK_SIZE] ? (A[innerI+HALF_BLOCK_SIZE][k] + B[k][innerJ+HALF_BLOCK_SIZE]) : C[innerI+HALF_BLOCK_SIZE][innerJ+HALF_BLOCK_SIZE];
    }

    dist[i*BLOCK_SIZE*n + j*BLOCK_SIZE + innerI*n + innerJ] = C[innerI][innerJ];
    dist[i*BLOCK_SIZE*n + j*BLOCK_SIZE + (innerI+HALF_BLOCK_SIZE)*n + innerJ] = C[innerI+HALF_BLOCK_SIZE][innerJ];
    dist[i*BLOCK_SIZE*n + j*BLOCK_SIZE + innerI*n + innerJ+HALF_BLOCK_SIZE] = C[innerI][innerJ+HALF_BLOCK_SIZE];
    dist[i*BLOCK_SIZE*n + j*BLOCK_SIZE + (innerI+HALF_BLOCK_SIZE)*n + innerJ+HALF_BLOCK_SIZE] = C[innerI+HALF_BLOCK_SIZE][innerJ+HALF_BLOCK_SIZE];
}

int main(int argc, char *argv[]){
	/******************************* load data *********************************/
    // only two arguments are allowed
    assert(argc == 3);

    int E, V;
    FILE *in_fp;
    in_fp = fopen(argv[1], "r");
    if(in_fp == NULL) printf("Failed on opening file\n");
    // read in data
    fread(&V, sizeof(int), 1, in_fp);
    fread(&E, sizeof(int), 1, in_fp);

    // compensate V to make V % BLOCK_SIZE == 0
	int comp_V = V + (BLOCK_SIZE - ((V-1) % BLOCK_SIZE + 1));

	//allocate memory
    int *adj_mat; 
    size_t sz = comp_V * comp_V * sizeof(int);
	hipHostMalloc((void**) &adj_mat, sz, hipHostMallocDefault);
	for(int i = 0; i < comp_V; i++){
        for(int j = 0; j < comp_V; j++){
            if(i == j) adj_mat[i*comp_V+j] = 0;
            else adj_mat[i*comp_V+j] = inf;
        }
    }
    // load data to graph
    int src, dst, w;
    while(E--){
        fread(&src, sizeof(int), 1, in_fp);
        fread(&dst, sizeof(int), 1, in_fp);
        fread(&w, sizeof(int), 1, in_fp);
        adj_mat[src*comp_V+dst] = w;
    }
    fclose(in_fp);
    /****************************************************************************/

    int *adj_mat_d[2];
    int round =  ceil((float) comp_V/BLOCK_SIZE);
    const int blocks = (comp_V + BLOCK_SIZE - 1) / BLOCK_SIZE;
	
    // 2D block
    dim3 threads(32, 32);
	dim3 p2(2, round-1);

	#pragma omp parallel num_threads(2)
	{
		int thread_id = omp_get_thread_num();
		hipSetDevice(thread_id);

        // Malloc memory
        hipMalloc(&adj_mat_d[thread_id], sz);

        // divide data
		int round_per_thd = round / 2;
		int y_offset = round_per_thd * thread_id;
        if(thread_id == 1) round_per_thd += round % 2;

		dim3 p3(round, round_per_thd);
		
        size_t cp_amount = comp_V * BLOCK_SIZE * round_per_thd * sizeof(int);
        hipMemcpy(adj_mat_d[thread_id] + y_offset *BLOCK_SIZE * comp_V, adj_mat + y_offset * BLOCK_SIZE * comp_V, cp_amount, hipMemcpyHostToDevice);

        size_t block_row_sz = BLOCK_SIZE * comp_V * sizeof(int);
		for(int r = 0; r < round; r++) {

            // Every thread has its own y_offset
            if (r >= y_offset && r < (y_offset + round_per_thd)) {
                hipMemcpy(adj_mat + r * BLOCK_SIZE * comp_V, adj_mat_d[thread_id] + r * BLOCK_SIZE * comp_V, block_row_sz, hipMemcpyDeviceToHost);
            }

            #pragma omp barrier
            hipMemcpy(adj_mat_d[thread_id] + r * BLOCK_SIZE * comp_V, adj_mat + r * BLOCK_SIZE * comp_V, block_row_sz, hipMemcpyHostToDevice);

            Phase_1 <<<1, threads>>>(adj_mat_d[thread_id], r, comp_V);
                
            //cudaDeviceSynchronize();

            Phase_2 <<<blocks, threads>>>(adj_mat_d[thread_id], r, comp_V);

            //cudaDeviceSynchronize();

            Phase_3 <<<p3, threads>>>(adj_mat_d[thread_id], r, comp_V, y_offset);
        }

		hipMemcpy(adj_mat + y_offset *BLOCK_SIZE * comp_V, adj_mat_d[thread_id] + y_offset *BLOCK_SIZE * comp_V, block_row_sz * round_per_thd, hipMemcpyDeviceToHost);
		#pragma omp barrier
	}
	
	// output
    FILE *out_fp;
    out_fp = fopen(argv[2], "wb");
    for(int i = 0; i < V; i++){
        for(int j = 0; j < V; j++){
            fwrite(adj_mat+i*comp_V+j, sizeof(int), 1, out_fp);
        }   
    }   
    fclose(out_fp);

	//free memory
	hipFree(adj_mat_d[0]);
    hipFree(adj_mat_d[1]);
    hipHostFree(adj_mat);
	return 0;
}