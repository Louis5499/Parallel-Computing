#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime.h>
#define BLOCK_SIZE 128 // TODO: 64

using namespace std;

const int INF = ((1 << 30) - 1);
// const int V = 50010;
void input(char* infile);
void output(char *outFileName);

void block_FW(int B);
int ceil(int a, int b);
__global__ void Phase1(int *dist, int Round, int n);
__global__ void Phase2(int *dist, int Round, int n);
__global__ void Phase3(int *dist, int Round, int n);

int original_n, n, m;
int* Dist = NULL;

int main(int argc, char* argv[]) {
	input(argv[1]);
	block_FW(BLOCK_SIZE);
	output(argv[2]);
    hipHostFree(Dist);
	return 0;
}

void input(char* infile) {
    cout << "input" << endl;
    FILE* file = fopen(infile, "rb");
    fread(&original_n, sizeof(int), 1, file);
    fread(&m, sizeof(int), 1, file);

    // make n % BLOCK_SIZE == 0
    n = original_n + (BLOCK_SIZE - (original_n%BLOCK_SIZE));

    Dist = (int*) malloc(sizeof(int)*n*n);

    for (int i = 0; i < n; ++ i) {
        for (int j = 0; j < n; ++ j) {
            if (i == j) {
                Dist[i*n+j] = 0;
            } else {
                Dist[i*n+j] = INF;
            }
        }
    }

    int pair[3];
    for (int i = 0; i < m; ++ i) {
        fread(pair, sizeof(int), 3, file);
        Dist[pair[0]*n+pair[1]] = pair[2];
    }
    fclose(file);

}

void output(char *outFileName) {
    FILE *outfile = fopen(outFileName, "w");
	for (int i = 0; i < original_n; ++i) {
		for (int j = 0; j < original_n; ++j) {
            if (Dist[i*n+j] >= INF) Dist[i*n+j] = INF;
        }
		fwrite(&Dist[i*n], sizeof(int), original_n, outfile);
	}
    fclose(outfile);
}

int ceil(int a, int b) { return (a + b - 1) / b; }

void block_FW(int B) {
    int* dst = NULL;

    const int matrixSize = n * n * sizeof(int);

    hipHostRegister(Dist, matrixSize, hipHostRegisterDefault);
    hipMalloc(&dst, matrixSize);
	hipMemcpy(dst, Dist, matrixSize, hipMemcpyHostToDevice);

    const int blocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 block_dim(BLOCK_SIZE, 1, 1); //TODO: 1024 = 32*32 threads
    dim3 grid_dim(blocks, blocks, 1);

    int round = ceil(n, B);
    for (int r = 0; r < round; ++r) {
        printf("%d %d\n", r, round);
        fflush(stdout);
        /* Phase 1*/
        Phase1<<<1, block_dim>>>(dst, r, n);

        /* Phase 2*/
        Phase2<<<blocks, block_dim>>>(dst, r, n);

        /* Phase 3*/
        Phase3<<<grid_dim, block_dim>>>(dst, r, n);
    }

    hipMemcpy(Dist, dst, matrixSize, hipMemcpyDeviceToHost);
	hipFree(dst);
}


inline __device__ void BlockCalc(int* C, int* A, int* B, int innerJ) {
    for (int k = 0; k < BLOCK_SIZE; k++) {
        for (int innerI=0; innerI < BLOCK_SIZE; innerI++) {
            int sum = A[innerI*BLOCK_SIZE + k] + B[k*BLOCK_SIZE + innerJ];
            if (C[innerI*BLOCK_SIZE + innerJ] > sum) {
                C[innerI*BLOCK_SIZE + innerJ] = sum;
            }
        }
        __syncthreads();
    }
    //   printf("New Added Element[%d][%d]: %d   Element[%d][%d]: %d  Combine Value: %d | Original Value: %d\n", bi, k, A[bi*BLOCK_SIZE + k], k, bj, B[k*BLOCK_SIZE + bj], sum, C[bi*BLOCK_SIZE + bj]);
  }

__global__ void Phase1(int *dist, int Round, int n) {
    // const int innerI = threadIdx.y;
    const int threadx = threadIdx.x;
    const int offset = BLOCK_SIZE * Round;

    __shared__ int A[BLOCK_SIZE]; // 1D Array unroll
    __shared__ int B[BLOCK_SIZE]; // 1D Array
    __shared__ int C[BLOCK_SIZE]; // 1D Array

    for (int k = 0; k < BLOCK_SIZE; k++) {
        A[threadx] = dist[offset*(n+1) + threadx*n + k];
        B[threadx] = dist[offset*(n+1) + k*n + threadx];
        for (int innerI=0; innerI < BLOCK_SIZE; innerI++) {
            C[threadx] = dist[offset*(n+1) + innerI*n + threadx];
            __syncthreads();
            int sum = A[innerI] + B[threadx];
            if (C[threadx] > sum) {
                C[threadx] = sum;
            }
            dist[offset*(n+1) + innerI*n + threadx] = C[threadx];
        }
        __syncthreads();
    }
}

__global__ void Phase2(int *dist, int Round, int n) {
    const int i = blockIdx.x; // "i" in n block in one row
    if (i == Round) return;

    // const int innerI = threadIdx.y;
    const int threadx = threadIdx.x;
    const int diagonalOffset = BLOCK_SIZE * Round;

    __shared__ int Diagonal_A[BLOCK_SIZE];
    __shared__ int Diagonal_B[BLOCK_SIZE];
    __shared__ int A[BLOCK_SIZE];
    __shared__ int A_RESULT[BLOCK_SIZE];
    __shared__ int B[BLOCK_SIZE];
    __shared__ int B_RESULT[BLOCK_SIZE];
  
    for (int k = 0; k < BLOCK_SIZE; k++) {
        A[threadx] = dist[i*BLOCK_SIZE*n + Round*BLOCK_SIZE + threadx*n + k];
        Diagonal_A[threadx] = dist[diagonalOffset*(n+1) + k*n + threadx];

        Diagonal_B[threadx] = dist[diagonalOffset*(n+1) + threadx*n + k];
        B[threadx] = dist[Round*BLOCK_SIZE*n + i*BLOCK_SIZE + k*n + threadx];
        
        for (int innerI=0; innerI < BLOCK_SIZE; innerI++) {
            A_RESULT[threadx] = dist[i*BLOCK_SIZE*n + Round*BLOCK_SIZE + innerI*n + threadx];
            B_RESULT[threadx] = dist[Round*BLOCK_SIZE*n + i*BLOCK_SIZE + innerI*n + threadx];
            __syncthreads();
            // BlockCalc(A, A, Diagonal, innerJ);
            int sum = A[innerI] + Diagonal_A[threadx];
            if (A_RESULT[threadx] > sum) {
                A_RESULT[threadx] = sum;
            }
            // BlockCalc(B, Diagonal, B, innerJ);
            sum = Diagonal_B[innerI] + B[threadx];
            if (B_RESULT[threadx] > sum) {
                B_RESULT[threadx] = sum;
            }
            
            dist[i*BLOCK_SIZE*n + Round*BLOCK_SIZE + innerI*n + threadx] = A_RESULT[threadx];
            dist[Round*BLOCK_SIZE*n + i*BLOCK_SIZE + innerI*n + threadx] = B_RESULT[threadx];
        }
        __syncthreads();
    }
}

__global__ void Phase3(int *dist, int Round, int n) {
    const int j = blockIdx.x;
    const int i = blockIdx.y;
    if (i == Round && j == Round) return;

    // const int innerI = threadIdx.y;
    const int threadx = threadIdx.x;

    __shared__ int A[BLOCK_SIZE];
    __shared__ int B[BLOCK_SIZE];
    __shared__ int C[BLOCK_SIZE];

    for (int k = 0; k < BLOCK_SIZE; k++) {
        A[threadx] = dist[i*BLOCK_SIZE*n + Round*BLOCK_SIZE + threadx*n + k];
        B[threadx] = dist[Round*BLOCK_SIZE*n + j*BLOCK_SIZE + k*n + threadx];
        for (int innerI=0; innerI < BLOCK_SIZE; innerI++) {
            C[threadx] = dist[i*BLOCK_SIZE*n + j*BLOCK_SIZE + innerI*n + threadx];
            __syncthreads();
            int sum = A[innerI] + B[threadx];
            if (C[threadx] > sum) {
                C[threadx] = sum;
            }
            dist[i*BLOCK_SIZE*n + j*BLOCK_SIZE + innerI*n + threadx] = C[threadx];
        }
        __syncthreads();
    }
  
    // for (int innerI=0; innerI < BLOCK_SIZE; innerI++) {
    //     C[innerI*BLOCK_SIZE + innerJ] = dist[i*BLOCK_SIZE*n + j*BLOCK_SIZE + innerI*n + innerJ];
    //     A[innerI*BLOCK_SIZE + innerJ] = dist[i*BLOCK_SIZE*n + Round*BLOCK_SIZE + innerI*n + innerJ];
    //     B[innerI*BLOCK_SIZE + innerJ] = dist[Round*BLOCK_SIZE*n + j*BLOCK_SIZE + innerI*n + innerJ];
    // }
  
    // __syncthreads();
  
    // BlockCalc(C, A, B, innerJ);
  
    // __syncthreads();
  
    // for (int innerI=0; innerI < BLOCK_SIZE; innerI++) dist[i*BLOCK_SIZE*n + j*BLOCK_SIZE + innerI*n + innerJ] = C[innerI*BLOCK_SIZE + innerJ];
}